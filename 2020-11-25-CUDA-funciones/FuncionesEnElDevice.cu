#include "hip/hip_runtime.h"
//Mi Primer Programa en CUDA
#include<iostream>
#include <fstream>
#include <cmath>
#include <GL/glew.h>
#include <GL/glut.h>
#include <hip/hip_runtime_api.h>
#include <cuda_gl_interop.h>
using namespace std;

#define Q 5

#define Lx 16
#define Nx 8
const int Mx=(Lx+Nx-1)/Nx;

//--------------- KERNELS ----------------
__constant__ float d_w[Q];
__constant__ int d_Vx[Q];
__constant__ int d_Vy[Q];

__device__ float SumeleUno(float x){
  return x+1;
}
__global__ void SumarleUnoATodos(float * d_a){
  int ix;
  ix=blockIdx.x*blockDim.x+threadIdx.x;
  d_a[ix]=SumeleUno(d_a[ix]);
}

int main(){
  int ix;
  //DECLARAR LAS MATRICES
  float h_w[Q]; int h_Vx[Q],h_Vy[Q];
  float h_a[Lx]; float*d_a; hipMalloc((void**) &d_a,Lx*sizeof(float));

  //INICIALIZAR LAS CONSTANTES Y LAS MANDO AL DEVICE
  //Cargarlos en el Host
  h_w[0]=1.0/3; h_w[1]=h_w[2]=h_w[3]=h_w[4]=1.0/6;
  h_Vx[0]=0;  h_Vx[1]=1;  h_Vx[2]=0;  h_Vx[3]=-1; h_Vx[4]=0;
  h_Vy[0]=0;  h_Vy[1]=0;  h_Vy[2]=1;  h_Vy[3]=0;  h_Vy[4]=-1;
  //Enviarlos al Device
  hipMemcpyToSymbol(HIP_SYMBOL(d_w),h_w,Q*sizeof(float),0,hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(d_Vx),h_Vx,Q*sizeof(int),0,hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(d_Vy),h_Vy,Q*sizeof(int),0,hipMemcpyHostToDevice);

  //INICIALIZAR LAS VARIABLES Y LAS MANDO AL DEVICE
  for(ix=0;ix<Lx;ix++) h_a[ix]=ix;
  hipMemcpy(d_a,h_a,Lx*sizeof(float),hipMemcpyHostToDevice);

  //PROCESAR EN LA TARJETA GRAFICA
  dim3 ThreadsPerBlock(Nx,1,1);
  dim3 BlocksPerGrid(Mx,1,1);
  SumarleUnoATodos<<<BlocksPerGrid,ThreadsPerBlock>>>(d_a);

  //IMPRIMIR LOS DATOS
  //Devolverlos al Host
  hipMemcpy(h_a,d_a,Lx*sizeof(float),hipMemcpyDeviceToHost);
  //Imprimirlos
  for(ix=0;ix<Lx;ix++) cout<<ix<<" "<<h_a[ix]<<endl;

  //LIBERAR MEMORIA
  hipFree(d_a);

  return 0;
}
