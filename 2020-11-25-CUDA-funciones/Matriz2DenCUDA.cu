#include "hip/hip_runtime.h"
//Mi Primer Programa en CUDA
#include<iostream>
#include <fstream>
#include <cmath>
#include <GL/glew.h>
#include <GL/glut.h>
#include <hip/hip_runtime_api.h>
#include <cuda_gl_interop.h>
using namespace std;

#define Q 5

#define Lx 16
#define Ly 8
#define Nx 8
#define Ny 8
const int Mx=(Lx+Nx-1)/Nx;
const int My=(Ly+Ny-1)/Ny;

//--------------- KERNELS ----------------
__constant__ float d_w[Q];
__constant__ int d_Vx[Q];
__constant__ int d_Vy[Q];

__device__ float SumeleUno(float x){
  return x+1;
}
__global__ void SumarleUnoATodos(float * d_a,size_t pitcha){
  int ix,iy; float *aux;
  ix=blockIdx.x*blockDim.x+threadIdx.x;
  iy=blockIdx.y*blockDim.y+threadIdx.y;

  aux=d_a+(ix*pitcha)/sizeof(float)+iy; // aux es &(d_a[ix][iy])
  
  (*aux)=SumeleUno(*aux); //  (*aux) es d_a[ix][iy]
}

int main(){
  int ix,iy;
  //DECLARAR LAS MATRICES
  float h_w[Q]; int h_Vx[Q],h_Vy[Q];

  //INICIALIZAR LAS CONSTANTES Y LAS MANDO AL DEVICE
  //Cargarlos en el Host
  h_w[0]=1.0/3; h_w[1]=h_w[2]=h_w[3]=h_w[4]=1.0/6;
  h_Vx[0]=0;  h_Vx[1]=1;  h_Vx[2]=0;  h_Vx[3]=-1; h_Vx[4]=0;
  h_Vy[0]=0;  h_Vy[1]=0;  h_Vy[2]=1;  h_Vy[3]=0;  h_Vy[4]=-1;
  //Enviarlos al Device
  hipMemcpyToSymbol(HIP_SYMBOL(d_w),h_w,Q*sizeof(float),0,hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(d_Vx),h_Vx,Q*sizeof(int),0,hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(d_Vy),h_Vy,Q*sizeof(int),0,hipMemcpyHostToDevice);

  //DECLARAR LAS VARIABLES
  //declarar en Host
  float h_a[Lx][Ly];
  //declarar en Device
  float*d_a; size_t pitcha;
  hipMallocPitch((void**) &d_a,&pitcha,Ly*sizeof(float),Lx);

  //INICIALIZAR LAS VARIABLES Y LAS MANDO AL DEVICE
  //Cargar los datos en el Host
  for(ix=0;ix<Lx;ix++)
    for(iy=0;iy<Ly;iy++)
      h_a[ix][iy]=Ly*ix+iy;
  //Mostrar
  for(ix=0;ix<Lx;ix++){
    for(iy=0;iy<Ly;iy++)
      cout<<h_a[ix][iy]<<" ";
    cout<<endl;
  }
  cout<<endl;
  //Enviarlos al Device
  hipMemcpy2D(d_a,pitcha,h_a,Ly*sizeof(float),Ly*sizeof(float),Lx,hipMemcpyHostToDevice);

  //PROCESAR EN LA TARJETA GRAFICA
  dim3 ThreadsPerBlock(Nx,Ny,1);
  dim3 BlocksPerGrid(Mx,My,1);
  SumarleUnoATodos<<<BlocksPerGrid,ThreadsPerBlock>>>(d_a,pitcha);

  //IMPRIMIR LOS DATOS
  //Devolverlos al Host
  hipMemcpy2D(h_a,Ly*sizeof(float),d_a,pitcha,Ly*sizeof(float),Lx,hipMemcpyDeviceToHost);
  //Mostrar
  for(ix=0;ix<Lx;ix++){
    for(iy=0;iy<Ly;iy++)
      cout<<h_a[ix][iy]<<" ";
    cout<<endl;
  }
  cout<<endl;

  //LIBERAR MEMORIA
  hipFree(d_a);

  return 0;
}
